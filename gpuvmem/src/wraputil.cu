#include "wraputil.cuh"

Synthesizer * wrapSynth(){
        enum {MFS}; // Synthesizer
        return (Singleton<SynthesizerFactory>::Instance().CreateSynthesizer(MFS));
}
Optimizator * wrapOpti(){
        enum {CG,LBFGS}; // Synthesizer
        return (Singleton<OptimizatorFactory>::Instance().CreateOptimizator(CG));
}
ObjectiveFunction * wrapOf(){
        enum {DefaultObjectiveFunction}; // ObjectiveFunction
        return (Singleton<ObjectiveFunctionFactory>::Instance().CreateObjectiveFunction(DefaultObjectiveFunction));
}
Io * wrapIo(){
        enum {MS}; // Io
        return (Singleton<IoFactory>::Instance().CreateIo(MS)); // This is the default Io Class
}
Fi * wrapFi(int id){
        enum {Chi2, Entropy, Laplacian, QuadraticPenalization, TotalVariation, TotalSquaredVariation, L1Norm};
        if(id==0)
                return(Singleton<FiFactory>::Instance().CreateFi(Chi2) );
        else if(id==1)                
                return (Singleton<FiFactory>::Instance().CreateFi(Entropy));
        else if(id==2)       
                return(Singleton<FiFactory>::Instance().CreateFi(Laplacian));
        else if(id==3)
                return (Singleton<FiFactory>::Instance().CreateFi(QuadraticPenalization));
        else if(id==4)
                return(Singleton<FiFactory>::Instance().CreateFi(QuadraticPenalization));
        else if(id==5)
                return(Singleton<FiFactory>::Instance().CreateFi(QuadraticPenalization));
        else if(id==6)
                return(Singleton<FiFactory>::Instance().CreateFi(QuadraticPenalization));
}
int checkRequirements(){
        hipGetDeviceCount(&num_gpus);

        printf("gpuvmem Copyright (C) 2016-2017  Miguel Carcamo, Pablo Roman, Simon Casassus, Victor Moral, Fernando Rannou, Nicolás Muñoz - miguel.carcamo@protonmail.com\n");
        printf("This program comes with ABSOLUTELY NO WARRANTY; for details use option -w\n");
        printf("This is free software, and you are welcome to redistribute it under certain conditions; use option -c for details.\n\n\n");


        if(num_gpus < 1) {
                printf("No CUDA capable devices were detected\n");
                return 1;
        }
}
